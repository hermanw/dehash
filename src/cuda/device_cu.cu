#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include "device_cu.h"

typedef unsigned char uchar;
typedef unsigned int uint;

#include "md5.inc"

void check(hipError_t e, int const line)
{
    if (e)
    {
        printf("hipError_t(%d) @%d\n", e, line);
        exit(e);
    }
}

#define check_error(val) check(val, __LINE__)

#define BLOCK_LEN 64 // In bytes
#define LENGTH_SIZE 8 // In bytes

struct GpuSection
{
    ds_type type;
    int index;
    int length;
};


__global__ void compute(
    uint* p_count,
    const uchar* p_input,
    uchar* p_output,
    const ulong2* p_hash,
    const uchar* p_number,
    const uchar* p_helper,
    int hash_len,
    int data_length,
    int gpu_section_size,
    GpuSection *gs
    )
{
    if(*p_count >= hash_len) return;

    // fill data
    uchar data[BLOCK_LEN]= {0};
    data[data_length] = 0x80;
    data[BLOCK_LEN - LENGTH_SIZE] = (uchar)(data_length << 3);
    for (int i = 0; i < data_length; i++) {
        data[i] = p_input[i];
    }

    for (int i = 0; i < gpu_section_size; i++)
    {
        uint index = 0;
        if(i==0) index = blockIdx.x;
        else if(i==1) index = blockIdx.y;
        else index = blockIdx.z;

        if(gs[i].type == ds_type_list)
        {
            uint offset = index * gs[i].length;
            for (uint i = 0; i < gs[i].length; i++)
            {
                data[gs[i].index + i] = p_helper[offset + i];
            }
        }
        else if(gs[i].type == ds_type_digit)
        {
            uint offset = (index << 2) + 4 - gs[i].length;
            for (uint i = 0; i < gs[i].length; i++)
            {
                data[gs[i].index + i] = p_number[offset + i];
            }
        }
        else if(gs[i].type == ds_type_idsum)
        {
            const uchar COE[17] = {7,9,10,5,8,4,2,1,6,3,7,9,10,5,8,4,2};
            const uchar C_SUM[11] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'X'};
            uint sum = 0;
            for (uint j = 0; j < 17; j++)
            {
                sum += (data[j] - '0') * COE[j];
            }
            uint r = sum % 11;
            r = 12 - r;
            r = r>10?r-11:r;
            data[gs[i].index] =C_SUM[r];
        }
    }

    // hash
    uint4 hash = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    md5(&hash, data);

    ulong2 bhash;
    bhash.x = hash.x << 32 + hash.y;
    bhash.y = hash.z << 32 + hash.w;
    int low = 0;
    int high = hash_len - 1;
    do
    {
        int mid = (low + high) / 2;
        ulong2 ahash = p_hash[mid];
        if(ahash.x < bhash.x)
        {
            low = mid + 1;
        }
        else if (ahash.x > bhash.x)
        {
            high = mid - 1;
        }
        else
        {
            if(ahash.y < bhash.y) {low = mid + 1;}
            else if (ahash.y > bhash.y) {high = mid - 1;}
            else
            {
                atomicInc(p_count, 1);
                int offset = mid * data_length;
                for (int j = 0; j < data_length; j++)
                {
                    p_output[offset + j] = data[j];
                }
                break;
            }
        }
    } while (low <= high);
}

void DeviceCu::init(Cfg &cfg)
{
    m_cfg = &cfg;
    check_error(hipSetDevice(m_device));
}

DeviceCu::~DeviceCu()
{
    hipFree(d_count_buffer);
    hipFree(d_input_buffer);
    hipFree(d_output_buffer);
    hipFree(d_hash_buffer);
    hipFree(d_number_buffer);
    hipFree(d_helper_buffer);
}

void DeviceCu::create_buffers(
                    int input_buffer_size,
                    void *p_hash, int hash_buffer_size, int hash_length,
                    void *p_number, int number_buffer_size,
                    void *p_helper, int helper_buffer_size)
{
    int count = 0;
    check_error(hipMalloc((void**)&d_count_buffer, sizeof(int)));
    check_error(hipMemcpy(d_count_buffer, &count, sizeof(int), hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_input_buffer, input_buffer_size));

    check_error(hipMalloc((void**)&d_output_buffer, hash_buffer_size * input_buffer_size));

    check_error(hipMalloc((void**)&d_hash_buffer, hash_buffer_size * hash_length));
    check_error(hipMemcpy(d_hash_buffer, p_hash, hash_buffer_size * hash_length, hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_number_buffer, number_buffer_size));
    check_error(hipMemcpy(d_number_buffer, p_number, number_buffer_size, hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_helper_buffer, helper_buffer_size));
    check_error(hipMemcpy(d_helper_buffer, p_helper, helper_buffer_size, hipMemcpyHostToDevice));
}

void DeviceCu::submit(void *p_input, int input_buffer_size, int hash_buffer_size)
{
    m_hash_buffer_size = hash_buffer_size;
    check_error(hipMemcpy(d_input_buffer, p_input, input_buffer_size, hipMemcpyHostToDevice));
    check_error(hipDeviceSynchronize());
}

int DeviceCu::run(size_t kernel_work_size[3])
{
    // TODO: need revisit
    dim3 numBlocks(kernel_work_size[0], kernel_work_size[1], kernel_work_size[2]);
    if (numBlocks.x > 100) numBlocks.x /= 100;
    else if (numBlocks.y > 100) numBlocks.y /= 100;
    else if (numBlocks.z > 100) numBlocks.z /= 100;

    int data_length = m_cfg->length;
    int gpu_section_size = m_cfg->gpu_sections.size();
    GpuSection gs[3];
    for(int i = 0; i < gpu_section_size; i++)
    {
        auto &ds = m_cfg->gpu_sections[i];
        gs[i].index = ds.index;
        gs[i].length = ds.length;
        gs[i].type = ds.type;
    }

    GpuSection *d_gs;
    check_error(hipMalloc((void**)&d_gs, sizeof(GpuSection)*3));
    check_error(hipMemcpy(d_gs, gs, sizeof(GpuSection)*3, hipMemcpyHostToDevice));

    compute<<<numBlocks, 100>>> ((uint*)d_count_buffer, 
                                    (const uchar*)d_input_buffer, 
                                    (uchar*)d_output_buffer, 
                                    (const ulong2*)d_hash_buffer, 
                                    (const uchar*)d_number_buffer, 
                                    (const uchar*)d_helper_buffer,
                                    m_hash_buffer_size, data_length, gpu_section_size, d_gs);
    
    check_error(hipDeviceSynchronize());
    hipFree(d_gs);
    return 0;
}

void DeviceCu::read_results(void* p_output, int length)
{
    check_error(hipMemcpy(p_output, d_output_buffer, length, hipMemcpyDeviceToHost));
    check_error(hipDeviceSynchronize());
}