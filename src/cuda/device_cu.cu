#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include "device_cu.h"

typedef unsigned char uchar;
typedef unsigned int uint;

#define DEVICE_FUNC_PREFIX __device__
#include "md5.inc"

void check(hipError_t e, int const line)
{
    if (e)
    {
        printf("hipError_t(%d) @%d\n", e, line);
        exit(e);
    }
}

#define check_error(val) check(val, __LINE__)

#define BLOCK_LEN 64 // In bytes
#define LENGTH_SIZE 8 // In bytes
#define THREAD_NUM 500

struct ULONG2
{
    unsigned long long x;
    unsigned long long y;
};

struct GpuSection
{
    ds_type type;
    int index;
    int length;
};

__global__ void compute(
    uint* p_count,
    const uchar* p_input,
    uchar* p_output,
    const ULONG2* p_hash,
    const uchar* p_number,
    const uchar* p_helper,
    int hash_len,
    int data_length,
    int gpu_section_size,
    GpuSection *gs
    )
{
    if(*p_count >= hash_len) return;

    // fill data
    uchar data[BLOCK_LEN]= {0};
    data[data_length] = 0x80;
    data[BLOCK_LEN - LENGTH_SIZE] = (uchar)(data_length << 3);
    memcpy(data, p_input, data_length);

    for (int i = 0; i < gpu_section_size; i++)
    {
        uint index = 0;
        // THIS IS A TRICKY: always break the 1st gpu section into blocks and threads
        if(i==0) index = blockIdx.x*THREAD_NUM + threadIdx.x;
        else if(i==1) index = blockIdx.y;
        else index = blockIdx.z;

        if(gs[i].type == ds_type_list)
        {
            uint offset = index * gs[i].length;
            memcpy(data+gs[i].index, p_helper+offset, gs[i].length);
        }
        else if(gs[i].type == ds_type_digit)
        {
            uint offset = (index << 2) + 4 - gs[i].length;
            memcpy(data+gs[i].index, p_number+offset, gs[i].length);
        }
        else if(gs[i].type == ds_type_idsum)
        {
            const uchar COE[17] = {7,9,10,5,8,4,2,1,6,3,7,9,10,5,8,4,2};
            const uchar C_SUM[11] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'X'};
            uint sum = 0;
            for (uint j = 0; j < 17; j++)
            {
                sum += (data[j] - '0') * COE[j];
            }
            uint r = sum % 11;
            r = 12 - r;
            r = r>10?r-11:r;
            data[gs[i].index] =C_SUM[r];
        }
    }


    // hash
    uint4 hash = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    md5(&hash, data);

    ULONG2 bhash;
    bhash.x = (unsigned long long)hash.x; bhash.x += (unsigned long long)hash.y << 32;
    bhash.y = (unsigned long long)hash.z; bhash.y += (unsigned long long)hash.w << 32;
    int low = 0;
    int high = hash_len - 1;
    do
    {
        int mid = (low + high) / 2;
        ULONG2 ahash = p_hash[mid];
        if(ahash.x < bhash.x)
        {
            low = mid + 1;
        }
        else if (ahash.x > bhash.x)
        {
            high = mid - 1;
        }
        else
        {
            if(ahash.y < bhash.y) {low = mid + 1;}
            else if (ahash.y > bhash.y) {high = mid - 1;}
            else
            {
                atomicAdd(p_count, 1);
                int offset = mid * data_length;
                for (int j = 0; j < data_length; j++)
                {
                    p_output[offset + j] = data[j];
                }
                break;
            }
        }
    } while (low <= high);
}

void DeviceCu::init(Cfg &cfg)
{
    m_cfg = &cfg;
    check_error(hipSetDevice(m_device));

    // kernel_work_size maps to 3 gpu sections
    // THIS IS A TRICKY: always break the 1st gpu section into blocks and threads
    numBlocks.x = m_cfg->kernel_work_size[0] / THREAD_NUM;
    numBlocks.y = m_cfg->kernel_work_size[1];
    numBlocks.z = m_cfg->kernel_work_size[2];
}

DeviceCu::~DeviceCu()
{
    hipFree(d_count_buffer);
    hipFree(d_input_buffer);
    hipFree(d_output_buffer);
    hipFree(d_hash_buffer);
    hipFree(d_number_buffer);
    hipFree(d_helper_buffer);
    hipFree(d_gs);
}

void DeviceCu::create_buffers(
                    int input_buffer_size,
                    void *p_hash, int hash_buffer_size, int hash_length,
                    void *p_number, int number_buffer_size,
                    void *p_helper, int helper_buffer_size)
{
    int count = 0;
    check_error(hipMalloc((void**)&d_count_buffer, sizeof(int)));
    check_error(hipMemcpy(d_count_buffer, &count, sizeof(int), hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_input_buffer, input_buffer_size));

    check_error(hipMalloc((void**)&d_output_buffer, hash_buffer_size * input_buffer_size));

    check_error(hipMalloc((void**)&d_hash_buffer, hash_buffer_size * hash_length));
    check_error(hipMemcpy(d_hash_buffer, p_hash, hash_buffer_size * hash_length, hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_number_buffer, number_buffer_size));
    check_error(hipMemcpy(d_number_buffer, p_number, number_buffer_size, hipMemcpyHostToDevice));

    check_error(hipMalloc((void**)&d_helper_buffer, helper_buffer_size));
    check_error(hipMemcpy(d_helper_buffer, p_helper, helper_buffer_size, hipMemcpyHostToDevice));

    int gpu_section_size = m_cfg->gpu_sections.size();
    GpuSection gs[3];
    for(int i = 0; i < gpu_section_size; i++)
    {
        auto &ds = m_cfg->gpu_sections[i];
        gs[i].index = ds.index;
        gs[i].length = ds.length;
        gs[i].type = ds.type;
    }
    check_error(hipMalloc((void**)&d_gs, sizeof(GpuSection)*3));
    check_error(hipMemcpy(d_gs, gs, sizeof(GpuSection)*3, hipMemcpyHostToDevice));
}

void DeviceCu::submit(void *p_input, int input_buffer_size, int hash_buffer_size)
{
    m_hash_buffer_size = hash_buffer_size;
    check_error(hipMemcpy(d_input_buffer, p_input, input_buffer_size, hipMemcpyHostToDevice));
    check_error(hipDeviceSynchronize());
}

int DeviceCu::run()
{
    int data_length = m_cfg->length;
    int gpu_section_size = m_cfg->gpu_sections.size();
    compute<<<numBlocks, THREAD_NUM>>>((uint *)d_count_buffer,
                                       (const uchar *)d_input_buffer,
                                       (uchar *)d_output_buffer,
                                       (const ULONG2 *)d_hash_buffer,
                                       (const uchar *)d_number_buffer,
                                       (const uchar *)d_helper_buffer,
                                       m_hash_buffer_size, data_length, gpu_section_size, (GpuSection *)d_gs);

    int count = 0;
    check_error(hipMemcpy(&count, d_count_buffer, sizeof(int), hipMemcpyDeviceToHost));
    check_error(hipDeviceSynchronize());
    return count;
}

void DeviceCu::read_results(void* p_output, int length)
{
    check_error(hipMemcpy(p_output, d_output_buffer, length, hipMemcpyDeviceToHost));
    check_error(hipDeviceSynchronize());
}