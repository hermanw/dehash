#include "enumerator_cu.h"
#include "device_cu.h"

void EnumeratorCu::enum_devices(std::vector<Device*> &list)
{
    int count = 0;
    if (hipGetDeviceCount(&count))
    {
        return;
    }

    for (int i = 0; i< count; i++)
    {
        hipDeviceProp_t deviceProps;
        if (hipGetDeviceProperties(&deviceProps, i) == hipSuccess)
        {
            DeviceCu *device = new DeviceCu();
            device->m_device = i;
            device->info += "name:";
            device->info += deviceProps.name;
            list.push_back(device);
        }
    }
}